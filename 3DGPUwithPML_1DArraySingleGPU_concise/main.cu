#include "hip/hip_runtime.h"
/***************************************************************************************/
/******************************************2019-01-05***********************************/
/************************************Author:Wenqiang Wang*******************************/
/**********************Southern University of Science and Technology********************/
/***************************************************************************************/
#include <iostream>
#include "ElasticWaveEquation.h"

int main()
{
	GRID grid = { 300,200, 100 };
	Delta delta = { 0.0005f, 10.0f, 10.0f, 10.0f };
	//Delta delta = { 0.00175f, 100.0f, 100.0f, 100.0f };
	//Delta delta = { 0.0005f, 5.0f, 5.0f, 5.0f };
	int pml = 10;

	SOURCELOCATION sourceLocation = { 0 };
	STATIONLOCATION stationLocation;	

	
	sourceLocation.x = grid.xLength / 2;
	sourceLocation.y = grid.yLength / 2;
	sourceLocation.z = grid.zLength / 2;

	//stationLocation.x = grid.xLength - 2 * pml;
	//stationLocation.y = grid.yLength - 2 * pml;
	//stationLocation.z = grid.zLength - 2 * pml;


	stationLocation.x = grid.xLength / 2 + pml;
	stationLocation.y = grid.yLength / 2 + pml;
	stationLocation.z = grid.zLength / 2 + pml;

	int timeLength = 1500;
	//float mainFrequncy = 2.0f;
	float mainFrequncy = 20.0f;

	DifferenceCoefficient dc;

	dc.diff_coef[0] = 1.21124f;
	dc.diff_coef[1] = -8.97217e-2f;
	dc.diff_coef[2] = 1.38428e-2f;
	dc.diff_coef[3] = -1.76566e-3f;
	dc.diff_coef[4] = 1.18680e-4f;

	Medium medCPU;

	ALLOCATE_3D_Medium( medCPU );

	
	for ( int i = 0; i < grid.xLength * grid.yLength * grid.zLength; i++)
	{
		medCPU.buoyancy[i] = 1.0f / 2600.0f;
		medCPU.lambda[i] = 2600.0f * pow( 5000.0f, 2);
		medCPU.mu[i] = 2600.0f * pow(5000.0f, 2) / 3.0f;
	}


	dim3 blocksPerGrid( ( grid.xLength + 31 ) / 32, ( grid.yLength + 15 ) / 16, grid.zLength );//the num of blocks in every grid
	dim3 threadsPerBlock( 32, 16 ); //the num of threads in every block
	GPUDim gpuDim = { blocksPerGrid, threadsPerBlock };

	time_t start, stop;
	start = time(NULL);
	// clock_t start, stop;
	// start = clock();
	// hipEvent_t start, stop;
	// hipEventCreate( &start );
	// hipEventCreate( &stop );
	// hipEventRecord( start );

	ElasticWaveEquation ewq = ElasticWaveEquation(
		timeLength, gpuDim, pml,
		grid, delta, medCPU,
		sourceLocation, stationLocation,
		dc, 
		mainFrequncy );

	ewq.run();
	stop = time(NULL);
	// stop = clock();
	cout << "Time Loss:" << (stop - start) << endl;

	// float elapseTime;
	//hipEventRecord( stop );
	// hipEventElapsedTime( &elapseTime, start, stop );
	// cout << "Time Loss:" << elapseTime << endl;

	// hipEventDestroy( start );
	// hipEventDestroy( stop );
	
	DELETE_3D_Medium( medCPU );

	//system("pause");
}
